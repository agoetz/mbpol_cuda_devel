
#include <hip/hip_runtime.h>
extern "C" __device__ double computeInteraction(
        const unsigned int atom1,
        const unsigned int atom2,
        const double4* __restrict__ posq,
        const double4* periodicBoxSize,
        double3 * forces) {

        // CUDA COMPUTATIONAL KERNEL

        return 0;
}

__global__ void evaluate_2b(
        const double4* __restrict__ posq,
        const double4* periodicBoxSize,
        double3 * forces,
        double * energy) {
        energy[0] = computeInteraction(0, 1, posq, periodicBoxSize, forces);
}

void launch_evaluate_2b(
        const double4* __restrict__ posq,
        const double4* periodicBoxSize,
        double3 * forces,
        double * energy) {
    evaluate_2b<<<1,1>>>(posq, periodicBoxSize, forces, energy);
    hipDeviceSynchronize();
}
